
#include <hip/hip_runtime.h>
__global__ void deriv_entropy(int n_train, int n_classes, 
			float* targets, float* sigma_o, float* d_entropy)
{
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int stride = blockDim.x;
	int idx;
	
	for(idx=bx*n_classes+tx; idx<n_train*n_classes; idx+=stride)
	{
		if(idx < n_train*n_classes)
			d_entropy[idx] = -targets[idx] / sigma_o[idx];
	}
}


