
#include <hip/hip_runtime.h>
__global__ void matmul(int n, const float *A, const float *B, float *C){

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int row = by*blockDim.y + ty;
  int col = bx*blockDim.x + tx;

  if(row < n && col < n){
    float val = 0.0;
    for(int i=0; i<n; ++i){
      val += A[row*n + i]*B[n*i + col];
    }
    C[row*n + col] = val;
  }
}


__global__ void addone(int n_cols, float *A)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	//int bx = blockIdx.x;
	//int by = blockIdx.y;
	
	int idx = tx*n_cols + ty;
	float val = tx*n_cols + ty + 1.0;
	//float val = bx*n_cols + by + 1.0;
	
	A[idx] = val;
}

